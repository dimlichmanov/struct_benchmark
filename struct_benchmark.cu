
#include <hip/hip_runtime.h>
#include <cstdio>
#include "omp.h"

#include <chrono>
#include <cmath>

#define ITER 10

//#define USE_NEC
#define USE_GPU
#define USE_FLOAT
//#define USE_VEC

#define SAFE_CALL( CallInstruction ) { \
cudaError_t cuerr = CallInstruction; \
cudaDeviceSynchronize();\
if(cuerr != hipSuccess) { \
printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
throw "error in CUDA API function, aborting..."; \
} \
}

#define SAFE_KERNEL_CALL( KernelCallInstruction ){ \
KernelCallInstruction; \
cudaError_t cuerr = hipGetLastError(); \
cudaDeviceSynchronize();\
if(cuerr != hipSuccess) { \
printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
throw "error in CUDA kernel launch, aborting..."; \
} \
if(cuerr != hipSuccess) { \
printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
throw "error in CUDA kernel execution, aborting..."; \
} \
}

struct vec {
    float x, y, z;
};


template<typename T>
__global__ void test_combined_bw_gpu(T *data1, T *data2, size_t length) {
    //(void) ftrace_region_begin("benchmark");
    int i = blockDim.x * blockIdx.x + threadIdx.x;

#ifdef USE_VEC

    data1[i].x = (data1[i].x - data2[i].x) / 2;
    data1[i].y = (data1[i].y - data2[i].y) / 2;
    data1[i].z = (data1[i].z - data2[i].z) / 2;

    //cudaDeviceSynchronize();

    data1[i].x = (data1[i].x - data2[i].x) / 2;
    data1[i].y = (data1[i].y - data2[i].y) / 2;
    data1[i].z = (data1[i].z - data2[i].z) / 2;

    //cudaDeviceSynchronize();

    data1[i].x = (data1[i].x - data2[i].x) / 2;
    data1[i].y = (data1[i].y - data2[i].y) / 2;
    data1[i].z = (data1[i].z - data2[i].z) / 2;

    //cudaDeviceSynchronize();

    data1[i].x = (data1[i].x - data2[i].x) / 2;
    data1[i].y = (data1[i].y - data2[i].y) / 2;
    data1[i].z = (data1[i].z - data2[i].z) / 2;

    //cudaDeviceSynchronize();

    data1[i].x = (data1[i].x - data2[i].x) / 2;
    data1[i].y = (data1[i].y - data2[i].y) / 2;
    data1[i].z = (data1[i].z - data2[i].z) / 2;
#endif

#ifdef USE_FLOAT

    data1[i] = (data1[i]- data2[i]) / 2;
    //cudaDeviceSynchronize();

    data1[i] = (data1[i]- data2[i]) / 2;
    //cudaDeviceSynchronize();

    data1[i] = (data1[i]- data2[i]) / 2;
    //cudaDeviceSynchronize();

    data1[i] = (data1[i]- data2[i]) / 2;
    //cudaDeviceSynchronize();

    data1[i] = (data1[i]- data2[i]) / 2;
    //cudaDeviceSynchronize();
#endif
//    (void) ftrace_region_end("benchmark");
}


template<typename T>
__global__ void test_x_bw(T *data1, T *data2) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    data1[i].x = data2[i].x;
}

template<typename T>
__global__ void test_xy_bw(T *data1, T *data2) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    data1[i].x = data2[i].x;
    data1[i].y = data2[i].y;
}

template<typename T>
__global__ void test_xyz_bw(T *data1, T *data2) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    data1[i].x = data2[i].x;
    data1[i].y = data2[i].y;
    data1[i].z = data2[i].z;
}

template<typename T>
void allocData(T **data, size_t length) {
#ifdef USE_NEC
    *data = (T *) aligned_alloc(sizeof(T), length * sizeof(T));
#endif
#ifdef USE_GPU
    SAFE_CALL(hipMalloc((void **) data, length * sizeof(T)));
#endif
}

template<typename T>
void freeData(T *data) {
#ifdef USE_NEC
    free(data);
#endif
#ifdef USE_GPU
    hipFree((void *) data);
#endif
}

int main(int argc, char **argv) {
    int coef = atoi(argv[1]);
    size_t length = 32 * 32 * 32 * coef;

#ifdef USE_FLOAT
    float *data1 = new float[length];
    float *data2 = new float[length];
    float *dev_data1;
    float *dev_data2;

    allocData<float>(&dev_data1, length);
    allocData<float>(&dev_data2, length);
#endif
#ifdef USE_VEC
    vec *data1 = new vec[length];
    vec *data2 = new vec[length];
    vec *dev_data1;
    vec *dev_data2;

    allocData<vec>(&dev_data1, length);
    allocData<vec>(&dev_data2, length);
#endif

#pragma omp parallel for
    for (int i = 0; i < length; i++) {
#ifdef USE_FLOAT
        data1[i] = 1.0f;
        data2[i] = 2.0f;
#endif
#ifdef USE_VEC
        data1[i].x = 1.0f;
        data2[i].x = 2.0f;
        data1[i].y = 1.0f;
        data2[i].y = 2.0f;
        data1[i].z = 1.0f;
        data2[i].z = 2.0f;
#endif
    }

#ifdef USE_FLOAT
    SAFE_CALL(hipMemcpy(dev_data1, data1, sizeof(float) * length, hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dev_data2, data2, sizeof(float) * length, hipMemcpyHostToDevice));
#endif
#ifdef USE_VEC
    SAFE_CALL(cudaMemcpy(dev_data1, data1, sizeof(vec) * length, cudaMemcpyHostToDevice));
    SAFE_CALL(cudaMemcpy(dev_data2, data2, sizeof(vec) * length, cudaMemcpyHostToDevice));
#endif

    int blockSize = 1024;
    int numBlocks = (length + blockSize - 1) / blockSize;
    double final_bw;

    for (int j = 0; j < ITER; j++) {
        std::chrono::high_resolution_clock::time_point tstart;
        tstart = std::chrono::high_resolution_clock::now();
        //test_bw_nec(dev_data1, dev_data2, length);
        //test_x_bw_nec(dev_data1, dev_data2, length);
        //test_xy_bw_nec(dev_data1, dev_data2, length);
        SAFE_KERNEL_CALL((test_combined_bw_gpu<<<numBlocks, blockSize>>>(dev_data1, dev_data2, length)));
        //test_xyz_bw_nec(dev_data1, dev_data2, length);
        std::chrono::high_resolution_clock::time_point tstop = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> time_span = tstop - tstart;
        double dt = time_span.count(); // in milliseconds

#ifdef USE_FLOAT
    double bw = 5 * 3 * 1000 * length * sizeof(float) / (dt * std::pow(10, 9));
#endif
#ifdef USE_VEC
    double bw = 5 * 3 * 1000 * length * sizeof(vec) / (dt * std::pow(10, 9));
#endif
        if (j == ITER - 1) {
            final_bw = bw;
        }
    }

    printf("  %ld\t%lf  ", length, final_bw);


#ifdef USE_FLOAT
    SAFE_CALL(hipMemcpy(data1, dev_data1, sizeof(float) * length, hipMemcpyDeviceToHost));
    SAFE_CALL(hipMemcpy(data2, dev_data2, sizeof(float) * length, hipMemcpyDeviceToHost));
#endif
#ifdef USE_VEC
    SAFE_CALL(cudaMemcpy(data1, dev_data1, sizeof(vec) * length, cudaMemcpyDeviceToHost));
    SAFE_CALL(cudaMemcpy(data2, dev_data2, sizeof(vec) * length, cudaMemcpyDeviceToHost));
#endif

#ifdef USE_FLOAT
    printf("%f", data1[102400]);
    printf("%f", data1[34]);
    printf("%f\n", data1[54]);
#endif
    #ifdef USE_VEC
    printf("%f", data1[102400].x);
    printf("%f", data1[34].y);
    printf("%f\n", data1[54].z);
    #endif

    freeData(dev_data1);
    freeData(dev_data2);
    delete[] data1;
    delete[] data2;
}
